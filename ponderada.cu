#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void sum_reduction(float* input, float* result, int n) {
    extern __shared__ float shared_data[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = (index < n) ? input[index] : 0.0f;
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        result[blockIdx.x] = shared_data[0];
    }
}

void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << message << " Error: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

int main() {
    int n = 1000000
    size_t size = n * sizeof(float);
    float* h_input = new float[n];
    float* h_result;

    for (int i = 0; i < n; i++) {
        h_input[i] = 1.0f;
    }

    float *d_input, *d_intermediate, *d_result;
    checkCudaError(hipMalloc(&d_input, size), "Falha ao alocar memória para d_input");
    checkCudaError(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice), "Falha ao copiar dados para d_input");

    int threads_per_block = 1024;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    checkCudaError(hipMalloc(&d_intermediate, blocks_per_grid * sizeof(float)), "Falha ao alocar memória para d_intermediate");
    checkCudaError(hipMalloc(&d_result, sizeof(float)), "Falha ao alocar memória para d_result");

    auto start = std::chrono::high_resolution_clock::now();
    sum_reduction<<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>(d_input, d_intermediate, n);
    hipDeviceSynchronize();
    h_result = new float[blocks_per_grid];
    checkCudaError(hipMemcpy(h_result, d_intermediate, blocks_per_grid * sizeof(float), hipMemcpyDeviceToHost), "Falha ao copiar resultados parciais para o host");

    float final_sum = 0.0f;
    for (int i = 0; i < blocks_per_grid; i++) {
        final_sum += h_result[i];
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    std::cout << "A soma total é: " << final_sum << std::endl;
    std::cout << "Tempo de execução (ms): " << duration.count() << " ms" << std::endl;

    delete[] h_input;
    delete[] h_result;
    hipFree(d_input);
    hipFree(d_intermediate);
    hipFree(d_result);

    return 0;
}
